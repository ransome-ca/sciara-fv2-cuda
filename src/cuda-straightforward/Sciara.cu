#include "hip/hip_runtime.h"
#include "Sciara.cuh"
#include "cal2DBuffer.cuh"

void allocateSubstates(Sciara *sciara)
{

  hipMallocManaged(&sciara->substates->Sz, sciara->domain->rows*sciara->domain->cols*sizeof(double));
  hipMallocManaged(&sciara->substates->Sz_next, sciara->domain->rows*sciara->domain->cols*sizeof(double));
  hipMallocManaged(&sciara->substates->Sh, sciara->domain->rows*sciara->domain->cols*sizeof(double));
  hipMallocManaged(&sciara->substates->Sh_next, sciara->domain->rows*sciara->domain->cols*sizeof(double));
  hipMallocManaged(&sciara->substates->ST, sciara->domain->rows*sciara->domain->cols*sizeof(double));
  hipMallocManaged(&sciara->substates->ST_next, sciara->domain->rows*sciara->domain->cols*sizeof(double));
  hipMallocManaged(&sciara->substates->Mf, sciara->domain->rows*sciara->domain->cols*NUMBER_OF_OUTFLOWS*sizeof(double));
  hipMallocManaged(&sciara->substates->Mb, sciara->domain->rows*sciara->domain->cols*sizeof(bool));
  hipMallocManaged(&sciara->substates->Mhs, sciara->domain->rows*sciara->domain->cols*sizeof(double));

  memset(sciara->substates->Sz,       0, sciara->domain->rows*sciara->domain->cols*sizeof(double));
  memset(sciara->substates->Sz_next,  0, sciara->domain->rows*sciara->domain->cols*sizeof(double));
  memset(sciara->substates->Sh,       0, sciara->domain->rows*sciara->domain->cols*sizeof(double));
  memset(sciara->substates->Sh_next,  0, sciara->domain->rows*sciara->domain->cols*sizeof(double));
  memset(sciara->substates->ST,       0, sciara->domain->rows*sciara->domain->cols*sizeof(double));
  memset(sciara->substates->ST_next,  0, sciara->domain->rows*sciara->domain->cols*sizeof(double));
  memset(sciara->substates->Mf,       0, sciara->domain->rows*sciara->domain->cols*NUMBER_OF_OUTFLOWS*sizeof(double));
  memset(sciara->substates->Mb,       0, sciara->domain->rows*sciara->domain->cols*sizeof(bool));
  memset(sciara->substates->Mhs,      0, sciara->domain->rows*sciara->domain->cols*sizeof(double));

}

void deallocateSubstates(Sciara *sciara)
{
// 	if(sciara->substates->Sz)       delete[] sciara->substates->Sz;
//   if(sciara->substates->Sz_next)  delete[] sciara->substates->Sz_next;
// 	if(sciara->substates->Sh)       delete[] sciara->substates->Sh;
//   if(sciara->substates->Sh_next)  delete[] sciara->substates->Sh_next;
// 	if(sciara->substates->ST)       delete[] sciara->substates->ST;
//   if(sciara->substates->ST_next)  delete[] sciara->substates->ST_next;
// 	if(sciara->substates->Mf)       delete[] sciara->substates->Mf;
// //if(sciara->substates->Mv)       delete[] sciara->substates->Mv;
// 	if(sciara->substates->Mb)       delete[] sciara->substates->Mb;
// 	if(sciara->substates->Mhs)      delete[] sciara->substates->Mhs;
}


void evaluatePowerLawParams(double PTvent, double PTsol, double value_sol, double value_vent, double &k1, double &k2)
{
	k2 = ( log10(value_vent) - log10(value_sol) ) / (PTvent - PTsol) ;
	k1 = log10(value_sol) - k2*(PTsol);
}

void simulationInitialize(Sciara* sciara)
{
  //dichiarazioni
  unsigned int maximum_number_of_emissions = 0;

  //azzeramento dello step dell'AC
  sciara->simulation->step = 0;
  sciara->simulation->elapsed_time = 0;

  //determinazione numero massimo di passi
  for (unsigned int i = 0; i < sciara->simulation->emission_rate.size(); i++)
    if (maximum_number_of_emissions < sciara->simulation->emission_rate[i].size())
      maximum_number_of_emissions = sciara->simulation->emission_rate[i].size();
  //maximum_steps_from_emissions = (int)(emission_time/Pclock*maximum_number_of_emissions);
  sciara->simulation->effusion_duration = sciara->simulation->emission_time * maximum_number_of_emissions;
  sciara->simulation->total_emitted_lava = 0;

  //definisce il bordo della morfologia
  MakeBorder(sciara);

  //calcolo a b (parametri viscosità) c d (parametri resistenza al taglio)
  evaluatePowerLawParams(
      sciara->parameters->PTvent, 
      sciara->parameters->PTsol, 
      sciara->parameters->Pr_Tsol,  
      sciara->parameters->Pr_Tvent,  
      sciara->parameters->a, 
      sciara->parameters->b);
  evaluatePowerLawParams(
      sciara->parameters->PTvent,
      sciara->parameters->PTsol,
      sciara->parameters->Phc_Tsol,
      sciara->parameters->Phc_Tvent,
      sciara->parameters->c,
      sciara->parameters->d);
}

int _Xi[] = {0, -1,  0,  0,  1, -1,  1,  1, -1}; // Xj: Moore neighborhood row coordinates (see below)
int _Xj[] = {0,  0, -1,  1,  0, -1, -1,  1,  1}; // Xj: Moore neighborhood col coordinates (see below)
void init(Sciara*& sciara)
{
  sciara = new Sciara;
  sciara->domain = new Domain;

  sciara->X = new NeighsRelativeCoords;

  hipMallocManaged(&sciara->X->Xi, MOORE_NEIGHBORS*sizeof(int));
  hipMallocManaged(&sciara->X->Xj, MOORE_NEIGHBORS*sizeof(int));

  for (int n=0; n<MOORE_NEIGHBORS; n++)
  {
    sciara->X->Xi[n] = _Xi[n];
    sciara->X->Xj[n] = _Xj[n];
  }

  sciara->substates = new Substates;
  //allocateSubstates(sciara); //Substates allocation is done when the confiugration is loaded
  sciara->parameters = new Parameters;
  sciara->simulation = new Simulation;
}

void finalize(Sciara*& sciara)
{
  deallocateSubstates(sciara);
  delete sciara->domain;
  // delete sciara->X->Xi;
  // delete sciara->X->Xj;
  delete sciara->X;
  delete sciara->substates;
  delete sciara->parameters;
  delete sciara->simulation;
  delete sciara;
  sciara = NULL;
}


void MakeBorder(Sciara *sciara) 
{
	int j, i;

	//prima riga
	i = 0;
	for (j = 0; j < sciara->domain->cols; j++)
		if (calGetMatrixElement(sciara->substates->Sz, sciara->domain->cols, i, j) >= 0)
			calSetMatrixElement(sciara->substates->Mb, sciara->domain->cols, i, j, true);

	//ultima riga
	i = sciara->domain->rows - 1;
	for (j = 0; j < sciara->domain->cols; j++)
		if (calGetMatrixElement(sciara->substates->Sz, sciara->domain->cols, i, j) >= 0)
			calSetMatrixElement(sciara->substates->Mb, sciara->domain->cols, i, j, true);

	//prima colonna
	j = 0;
	for (i = 0; i < sciara->domain->rows; i++)
		if (calGetMatrixElement(sciara->substates->Sz, sciara->domain->cols, i, j) >= 0)
			calSetMatrixElement(sciara->substates->Mb, sciara->domain->cols, i, j, true);
  
	//ultima colonna
	j = sciara->domain->cols - 1;
	for (i = 0; i < sciara->domain->rows; i++)
		if (calGetMatrixElement(sciara->substates->Sz, sciara->domain->cols, i, j) >= 0)
			calSetMatrixElement(sciara->substates->Mb, sciara->domain->cols, i, j, true);
	
	//il resto
	for (int i = 1; i < sciara->domain->rows - 1; i++)
		for (int j = 1; j < sciara->domain->cols - 1; j++)
			if (calGetMatrixElement(sciara->substates->Sz, sciara->domain->cols, i, j) >= 0) {
				for (int k = 1; k < MOORE_NEIGHBORS; k++)
					if (calGetMatrixElement(sciara->substates->Sz, sciara->domain->cols, i+sciara->X->Xi[k], j+sciara->X->Xj[k]) < 0)
          {
			      calSetMatrixElement(sciara->substates->Mb, sciara->domain->cols, i, j, true);
						break;
					}
			}
}
